#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256


__global__ void cuda_kernel(double *Cx,double *Cy,double *Cz,double *mass, double *ax, double *ay, double *az, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        ax[i] = 0.0;
        ay[i] = 0.0;
        az[i] = 0.0;

        for (int j = 0; j < N; j++) {
            if (i != j) {
                // Calcular distancia a la estrella
                double dx = Cx[i] - Cx[j];
                double dy = Cy[i] - Cy[j];
                double dz = Cz[i] - Cz[j];
                double dist_sq = dx * dx + dy * dy + dz * dz;
                double inv_dist = rsqrt(dist_sq);

                // Calcular fuerza aplicada a la estrella
                double force = -G * mass[j] * inv_dist * inv_dist * inv_dist;
                ax[i] fma(force, dx, ax[i]);
                ay[i] fma(force, dy, ay[i]);
                az[i] fma(force, dz, az[i]);
            }
        }
    }
}


extern "C" void compute_aceleration_CUDA(Star *stars, double *ax, double *ay, double *az, int N) {
    double *d_Cx, *d_Cy, *d_Cz, *d_mass;
    double *d_ax, *d_ay, *d_az;

    hipMalloc(&d_Cx, N * sizeof(double));
    hipMalloc(&d_Cy, N * sizeof(double));
    hipMalloc(&d_Cz, N * sizeof(double));
    hipMalloc(&d_mass, N * sizeof(double));
    hipMalloc(&d_ax, N * sizeof(double));
    hipMalloc(&d_ay, N * sizeof(double));
    hipMalloc(&d_az, N * sizeof(double));

    hipMemcpy(d_Cx, stars->Cx, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Cy, stars->Cy, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Cz, stars->Cz, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, stars->mass, N * sizeof(double), hipMemcpyHostToDevice);

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_kernel<<<numBlocks, BLOCK_SIZE>>>(d_Cx, d_Cy, d_Cz, d_mass, d_ax, d_ay, d_az, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        return;
    }

    hipMemcpy(ax, d_ax, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ay, d_ay, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(az, d_az, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_Cx);
    hipFree(d_Cy);
    hipFree(d_Cz);
    hipFree(d_mass);
    hipFree(d_ax);
    hipFree(d_ay);
    hipFree(d_az);
}



