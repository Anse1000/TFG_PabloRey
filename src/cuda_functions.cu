#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256


__global__ void cuda_kernel(const Star *estrellas, double *ax, double *ay, double *az, int N) {
    __shared__ Star sh_stars[BLOCK_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    double ax_i = 0.0, ay_i = 0.0, az_i = 0.0;
    double Ci_x = estrellas[i].C[0];
    double Ci_y = estrellas[i].C[1];
    double Ci_z = estrellas[i].C[2];

    for (int tile = 0; tile < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; tile++) {
        int j = tile * BLOCK_SIZE + threadIdx.x;
        if (j < N) {
            sh_stars[threadIdx.x] = estrellas[j];
        }
        __syncthreads();

        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; k++) {
            int j_global = tile * BLOCK_SIZE + k;
            if (j_global >= N || j_global == i) continue;

            double dx = sh_stars[k].C[0] - Ci_x;
            double dy = sh_stars[k].C[1] - Ci_y;
            double dz = sh_stars[k].C[2] - Ci_z;
            double dist_sq = dx * dx + dy * dy + dz * dz;

            double inv_dist = rsqrt(dist_sq);
            double force = -G * sh_stars[k].mass * inv_dist * inv_dist * inv_dist;

            ax_i += force * dx;
            ay_i += force * dy;
            az_i += force * dz;
        }
    }
    ax[i] = ax_i;
    ay[i] = ay_i;
    az[i] = az_i;
}

extern "C" void compute_aceleration_CUDA(Star *estrellas, double *ax, double *ay, double *az, int N) {
    Star *d_estrellas;
    double *d_ax, *d_ay, *d_az;

    hipMalloc(&d_estrellas, N * sizeof(Star));
    hipMalloc(&d_ax, N * sizeof(double));
    hipMalloc(&d_ay, N * sizeof(double));
    hipMalloc(&d_az, N * sizeof(double));

    hipMemcpy(d_estrellas, estrellas, N * sizeof(Star), hipMemcpyHostToDevice);

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_kernel<<<numBlocks, BLOCK_SIZE>>>(d_estrellas, d_ax, d_ay, d_az, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        return;
    }

    hipMemcpy(ax, d_ax, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ay, d_ay, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(az, d_az, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_estrellas);
    hipFree(d_ax);
    hipFree(d_ay);
    hipFree(d_az);
}


