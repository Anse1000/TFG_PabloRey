#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void cuda_kernel(double *Cx,double *Cy,double *Cz,double *mass, double *ax, double *ay, double *az, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        ax[i] = 0.0;
        ay[i] = 0.0;
        az[i] = 0.0;

        for (int j = 0; j < N; j++) {
            if (i != j) {
                // Calcular distancia a la estrella
                double dx = Cx[i] - Cx[j];
                double dy = Cy[i] - Cy[j];
                double dz = Cz[i] - Cz[j];
                double dist_sq = dx * dx + dy * dy + dz * dz;
                double inv_dist = rsqrt(dist_sq);
                // Calcular fuerza aplicada a la estrella
                double force = -G * mass[j] * inv_dist * inv_dist * inv_dist;
                ax[i] = fma(force, dx, ax[i]);
                ay[i] = fma(force, dy, ay[i]);
                az[i] = fma(force, dz, az[i]);
            }
        }
    }
}


extern "C" void compute_aceleration_CUDA(Star *stars, double *ax, double *ay, double *az, int N) {
    double *d_Cx, *d_Cy, *d_Cz, *d_mass;
    double *d_ax, *d_ay, *d_az;
	const size_t size = N * sizeof(double);


	// Usar streams para solapar transferencias
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipMalloc(&d_Cx, size);
    hipMalloc(&d_Cy, size);
    hipMalloc(&d_Cz, size);
    hipMalloc(&d_mass, size);
    hipMalloc(&d_ax, size);
    hipMalloc(&d_ay, size);
    hipMalloc(&d_az, size);

	// Transferir datos usando streams
    hipMemcpyAsync(d_Cx, stars->Cx, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_Cy, stars->Cy, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_Cz, stars->Cz, size, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(d_mass, stars->mass, size, hipMemcpyHostToDevice, stream2);

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    cuda_kernel<<<numBlocks, BLOCK_SIZE>>>(d_Cx, d_Cy, d_Cz, d_mass, d_ax, d_ay, d_az, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        return;
    }

	// Transferir resultados de vuelta
    hipMemcpyAsync(ax, d_ax, size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(ay, d_ay, size, hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(az, d_az, size, hipMemcpyDeviceToHost, stream2);

    // Sincronizar y limpiar
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);


    hipFree(d_Cx);
    hipFree(d_Cy);
    hipFree(d_Cz);
    hipFree(d_mass);
    hipFree(d_ax);
    hipFree(d_ay);
    hipFree(d_az);
}



